#include "hip/hip_runtime.h"
//#ifndef __HIPCC__  
//    #define __HIPCC__
//#endif
#include "hip/hip_runtime.h"
#include ""

#include "cuda_renderer.cuh"
#include "Vector3D.cuh"
#include "Matrix.cuh"
#include "Color.cuh"
#include "Camera.cuh"
#include "IGeometry.cuh"
#include "Plane.cuh"
#include "Node.cuh"
#include "IShader.cuh"
#include "Lambert.cuh"
#include "Sphere.cuh"
#include "OrenNayar.cuh"
#include "Phong.cuh"
#include "Refraction.cuh"
#include "Transform.cuh"
#include "Reflection.cuh"
#include "Layered.cuh"
#include "Fresnel.cuh"
#include "CameraController.cuh"
#include "RaytracerControls.cuh"
#include "Settings.cuh"
#include "WaterWaves.cuh"
#include "Scene.cuh"

__device__
bool needsAA[VFB_MAX_SIZE * VFB_MAX_SIZE];

__device__ 
Camera* dev_cam;
 
__device__
CameraController* controller;

__device__ Scene* scene;

__device__
bool testVisibility(const Vector& from, const Vector& to)
{
	Ray ray;
	ray.start = from;
	ray.dir = to - from;
	ray.dir.normalize();

	IntersectionData temp;
	temp.dist = (to - from).length();

	for (int i = 0; i < scene->dev_nodes.size(); ++i)
	{
		if (scene->dev_nodes[i]->intersect(ray, temp))
		{
			return false;
		}
	}

	return true;
}

__device__
Node* createNode(Geometry* geom, Shader* shader, Texture* tex = nullptr)
{
	scene->dev_geom.push_back(geom);
	scene->dev_shaders.push_back(shader);
	scene->dev_textures.push_back(tex);

	Node* node = new Node(geom, shader, tex);
	scene->dev_nodes.push_back(node);

	return node;
}

__global__ 
void initializeScene(bool realTime, int RES_X, int RES_Y)
{	
	scene = new Scene;
	scene->dev_lights.push_back(new RectLight(Vector(0, 296, 200), Vector(0, 0, 0), Vector(50, 34, 34), Color(1, 1, 1), 20, 6, 6));
	//scene->dev_lights.push_back(new PointLight(Vector(0, 296, 200), Color(1, 1, 1), 50000));
	scene->dev_lights[0]->beginFrame();

	dev_cam = new Camera;
	dev_cam->yaw = 0;
	dev_cam->pitch = 0;
	dev_cam->roll = 0;
	dev_cam->fov = 90;
	dev_cam->aspect = static_cast<float>(RES_X) / RES_Y;
	dev_cam->pos = Vector(0, 150, -100);
	dev_cam->beginFrame();

	controller = new CameraController(*dev_cam, 10.f);
	
	/*lightPos = Vector(0, 296, 100);
	lightColor = Color(1, 1, 1);
	lightPower = 60000;
	ambientLight = Color(0.2, 0.2, 0.2);*/

	if (realTime)
	{
		//createNode(new Plane(5), new Lambert(Color(0.5, 0.5, 0.5)));
		////createNode(new Plane(500), new OrenNayar(Color(0.5, 0.5, 0.5), 1.0));
		//createNode(new Sphere(Vector(0, 50, 200), 40.0), new Phong(Color(0, 0, 1), 32));

		// ocean
		createNode(new Plane(-30), new Lambert(Color(0x0AB6FF)));  // 0.1448, 0.4742, 0.6804   0x0AB6FF
		Layered* water = new Layered;
		water->addLayer(new Refraction(Color(0.9, 0.9, 0.9), 1.33), Color(1.0, 1.0, 1.0));
		water->addLayer(new Reflection(Color(0.9, 0.9, 0.9)), Color(1.0, 1.0, 1.0), new Fresnel(1.33));
	
		Node* waterGeom = createNode(new Plane(0), water, new WaterWaves(0.2));
		waterGeom->transform.scale(5, 5, 5);

		createNode(new Sphere(Vector(10, -20, 250), 100.0), new Lambert(Color(0, 1, 0)));
	}
	else
	{
		createNode(new Plane(5, 300, 300), new Lambert(Color(0xF5E08C)));

		Layered* mirror = new Layered;
		mirror->addLayer(new Reflection(), Color(1, 1, 1), new Fresnel(10.0));

		Node* BackWall = createNode(new Plane(-300, 300, 300), new Lambert(Color(0xF5E08C)));
		BackWall->transform.rotate(0, 90, 0);
	
		Node* SideWallLeft = createNode(new Plane(-150, 300, 300), new Lambert(Color(1.0, 0.0, 0.0)));
		SideWallLeft->transform.rotate(0, 0, 90);

		Node* SideWallRight = createNode(new Plane(150, 300, 300), new Lambert(Color(0.0, 0.0, 1.0)));
		SideWallRight->transform.rotate(0, 0, 90);

		Node* Roof = createNode(new Plane(300, 300, 300), new Lambert(Color(0xF5E08C)));

		Layered* moreGlossy = new Layered;
		moreGlossy->addLayer(new Phong(Color(0.0, 0.0, 1.0), 32), Color(1.0, 1.0, 1.0)); 
		moreGlossy->addLayer(new Reflection(Color(1.0, 1.0, 1.0)), Color(1, 1, 1), new Fresnel(2.5));
		createNode(new Sphere(Vector(0, 50, 200), 40.0), moreGlossy);

		Node* rectMirror = createNode(new Plane(0, 60, 80), mirror);
		rectMirror->transform.rotate(0, 90, 0);
		rectMirror->transform.translate(Vector(0, 120, 298));


		/// room
		/*createNode(new Plane(5, 300, 300), new OrenNayar(Color(0xF5E08C), 1.0));

		Layered* mirror = new Layered;
		mirror->addLayer(new Reflection(), Color(1, 1, 1), new Fresnel(10.0));

		Node* BackWall = createNode(new Plane(-300, 300, 300), new OrenNayar(Color(0xF5E08C), 1.0));
		BackWall->transform.rotate(0, 90, 0);
	
		Node* SideWallLeft = createNode(new Plane(-150, 300, 300), new OrenNayar(Color(1.0, 0.0, 0.0), 1.0));
		SideWallLeft->transform.rotate(0, 0, 90);

		Node* SideWallRight = createNode(new Plane(150, 300, 300), new OrenNayar(Color(0.0, 0.0, 1.0), 1.0));
		SideWallRight->transform.rotate(0, 0, 90);

		Node* Roof = createNode(new Plane(300, 300, 300), new OrenNayar(Color(0xF5E08C), 1.0));

		Layered* moreGlossy = new Layered;
		moreGlossy->addLayer(new Phong(Color(0.0, 0.0, 1.0), 32), Color(1.0, 1.0, 1.0)); 
		moreGlossy->addLayer(new Reflection(Color(1.0, 1.0, 1.0)), Color(1, 1, 1), new Fresnel(2.5));
		createNode(new Sphere(Vector(0, 50, 200), 40.0), moreGlossy);

		Node* rectMirror = createNode(new Plane(0, 60, 80), mirror);
		rectMirror->transform.rotate(0, 90, 0);
		rectMirror->transform.translate(Vector(0, 120, 298));*/

		/// ocean
		//createNode(new Plane(-300), new Lambert(Color(0x0AB6FF))); 
		//Layered* water = new Layered;
		//water->addLayer(new Refraction(Color(0.9, 0.9, 0.9), 1.33), Color(1.0, 1.0, 1.0));
		//water->addLayer(new Reflection(Color(0.9, 0.9, 0.9)), Color(1.0, 1.0, 1.0), new Fresnel(1.33));
		//
		//Node* waterGeom = createNode(new Plane(0), water, new WaterWaves(0.2));
		//waterGeom->transform.scale(5, 5, 5);

		//createNode(new Sphere(Vector(50, -20, 350), 100.0), new Lambert(Color(0, 1, 0)));
	}
}

__device__ 
Color raytrace(Ray ray)
{
	IntersectionData data;
	Node* closestNode = nullptr;

	if (ray.depth > MAX_RAY_DEPTH)
	{
		return Color(0, 0, 0);
	}

	data.dist = 1e99;

	for (int i = 0; i < scene->dev_nodes.size(); ++i)
	{
		if (scene->dev_nodes[i]->intersect(ray, data))
		{
			closestNode = scene->dev_nodes[i];
		}
	}

	// check if the closest intersection point is actually a light:
	bool hitLight = false;
	Color hitLightColor;
	for (int i = 0; i < scene->dev_lights.size(); ++i)
	{
		if (scene->dev_lights[i]->intersect(ray, data.dist))
		{
			hitLight = true;
			hitLightColor = scene->dev_lights[i]->getColor();
		}
	}
	if (hitLight) return hitLightColor;

	if (!closestNode)
	{
		//return Color(0, 0, 0);
		return Color(0.55f, 0.8f, 0.95f); // skyblue
		//return Color(1, 1, 1);
	}

	if (closestNode->bumpTex != nullptr)
	{
		closestNode->bumpTex->modifyNormal(data);
	}
	
	return closestNode->shader->shade(ray, data);
}

/**
 * @brief - The function checks if one of the red, green or blue components
 * of the colors a and b are too different.
 * @return true - if the difference is bigger than the THRESHOLD
 * @return false - if the difference is lower than the THRESHOLD
*/
__device__
inline bool tooDifferent(const Color& a, const Color& b)
{
	/*const float THRESHOLD = 0.1;
	return (fabs(a.r - b.r) > THRESHOLD ||
		     fabs(a.g - b.g) > THRESHOLD ||
		     fabs(a.b - b.b) > THRESHOLD);*/
	const float THRESHOLD = 0.1; // max color threshold; if met on any of the three channels, consider the colors too different
	for (int comp = 0; comp < 3; comp++) {
		float theMax = dev_max(a[comp], b[comp]);
		float theMin = dev_min(a[comp], b[comp]);

		// compare a single channel of the two colors. If the difference between them is large,
		// but they aren't overexposed, the difference will be visible: needs anti-aliasing.
		if (theMax - theMin > THRESHOLD && theMin < 1.33f) 
			return true;
	}
	return false;
}

__global__
void antiAliasing(Color* dev_vfb, bool previewAA, int RES_X, int RES_Y)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;	
		
	const int n_size = 5;
	Color neighs[n_size];
	neighs[0] = dev_vfb[offset];
	neighs[1] = dev_vfb[(x > 0 ? x - 1 : x) + y * blockDim.x * gridDim.x];
	neighs[2] = dev_vfb[(x + 1 < RES_X ? x + 1 : x) + y * blockDim.x * gridDim.x];
	neighs[3] = dev_vfb[x + (y > 0 ? y - 1 : y) * blockDim.x * gridDim.x];
	neighs[4] = dev_vfb[x + (y + 1 < RES_Y ? y + 1 : y) * blockDim.x * gridDim.x];

	Color average(0, 0, 0);
			
	for (int i = 0; i < n_size; ++i)
	{
		average += neighs[i];
	}
	average /= static_cast<float>(n_size);
			
	for (int i = 0; i < n_size; ++i)
	{
		if (tooDifferent(neighs[i], average))
		{
			needsAA[offset] = true;
			break;
		}
	}

	const double kernel[5][2] = {
			{ 0, 0 },
			{ 0.3, 0.3 },
			{ 0.6, 0 },
			{ 0, 0.6 },
			{ 0.6, 0.6 },
		};

	if (previewAA)
	{
		if (needsAA[offset])
		{
			dev_vfb[offset] = Color(1, 0, 0);
		}
	}
	else
	{
		if (needsAA[offset])
		{
			Color result = dev_vfb[offset];
			
			for (int i = 1; i < n_size; ++i)
			{
				result += raytrace(dev_cam->getScreenRay(x + kernel[i][0], y + kernel[i][1], RES_X, RES_Y));
			}
			dev_vfb[offset] = result / static_cast<float>(n_size);
		}
	}
}

__global__ 
void renderScene(Color* dev_vfb, int RES_X, int RES_Y)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < RES_X * RES_Y)
	{
		dev_vfb[offset] = raytrace(dev_cam->getScreenRay(x, y, RES_X, RES_Y));
	}
}

//__global__ 
//void renderScene2(Color* dev_vfb, int RES_X, int RES_Y)
//{
//	// map from threadIdx/BlockIdx to pixel position
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	if (offset >= RES_X * RES_Y / 2)
//	{
//		dev_vfb[offset] = raytrace(dev_cam->getScreenRay(x, y, RES_X, RES_Y));
//	}
//}

__global__
void freeMemory()
{
	delete dev_cam;
	delete controller;
	delete scene;
}

/**
 * Wrapper kernel function
*/
extern "C"
void initScene()
{
	initializeScene<<<1, 1>>>(GlobalSettings::realTime, GlobalSettings::RES_X, GlobalSettings::RES_Y);
}

__global__
void camBeginFrame()
{
	dev_cam->beginFrame();
}

extern "C"
void cameraBeginFrame()
{
	camBeginFrame<<<1, 1>>>();
}

extern "C" 
void cudaRenderer(Color* dev_vfb)
{
	dim3 THREADS_PER_BLOCK(32, 32); // 32*32 = 1024 (max threads per block supported)
	dim3 BLOCKS(GlobalSettings::RES_X / THREADS_PER_BLOCK.x, GlobalSettings::RES_Y / THREADS_PER_BLOCK.y); 
	
	// first pass
	renderScene<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_vfb, GlobalSettings::RES_X, GlobalSettings::RES_Y);

	//renderScene2<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_vfb, GlobalSettings::RES_X, GlobalSettings::RES_Y);

	//second pass
	if (GlobalSettings::AAEnabled)
	{
		antiAliasing<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_vfb, GlobalSettings::previewAA, GlobalSettings::RES_X, GlobalSettings::RES_Y);
	}
}

extern "C"
void freeDeviceMemory()
{	
	freeMemory<<<1, 1>>>();
}