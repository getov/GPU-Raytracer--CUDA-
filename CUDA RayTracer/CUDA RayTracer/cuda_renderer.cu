#include "hip/hip_runtime.h"
//#ifndef __HIPCC__  
//    #define __HIPCC__
//#endif
#include "hip/hip_runtime.h"
#include ""

#include "cuda_renderer.cuh"
#include "Vector3D.cuh"
#include "Matrix.cuh"
#include "Color.cuh"
#include "Camera.cuh"
#include "IGeometry.cuh"
#include "Plane.cuh"
#include "Node.cuh"
#include "IShader.cuh"
#include "Lambert.cuh"
#include "Sphere.cuh"
#include "OrenNayar.cuh"

__device__
bool needsAA[RES_X * RES_Y];

__device__ 
bool testVisibility(Node** dev_nodes, const IntersectionData& data)
{
	//Vector to = lightPos;
	// Vector from = ray.start
	Ray ray;
	ray.start = data.p + data.normal * 1e-3;

	ray.dir = lightPos - ray.start;
	ray.dir.normalize();
	
	IntersectionData temp;
	temp.dist = (lightPos - ray.start).length();
	
	for (int i = 0; i < GEOM_COUNT; ++i)
	{
		if (dev_nodes[i]->geom->intersect(ray, temp))
		{
			return false;
		}
	}

	return true;
}

__device__
void createNode(Geometry* geom, Shader* shader,
				Geometry** dev_geom, Shader** dev_shaders, Node** dev_nodes)
{
	if (GEOM_COUNT >= GEOM_MAX_SIZE)
	{
		return;
	}

	dev_geom[GEOM_COUNT]    = geom;
	dev_shaders[GEOM_COUNT] = shader;
	dev_nodes[GEOM_COUNT]   = new Node(dev_geom[GEOM_COUNT], dev_shaders[GEOM_COUNT]);
	++GEOM_COUNT;
}

__global__ 
void initializeScene(Camera* dev_cam, Geometry** dev_geom, Shader** dev_shaders, Node** dev_nodes)
{	
	dev_cam->yaw = 0;
	dev_cam->pitch = -30;
	dev_cam->roll = 0;
	dev_cam->fov = 90;
	dev_cam->aspect = 4.0 / 3.0;
	dev_cam->pos = Vector(0, 160, -50);

	dev_cam->beginFrame();
	
	lightPos = Vector(-90, 200, 150);
	lightColor = Color(1, 1, 1);
	lightPower = 50000;
	ambientLight = Color(0.2, 0.2, 0.2);

	createNode(new Plane(5), new OrenNayar(Color(0.0, 1.0, 0.0), 1.0),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(-150, 40, 180), 20.0), new Lambert(Color(1.0, 1.0, 0.0)),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(-50, 40, 180), 20.0), new Lambert(Color(0.5, 0.5, 0.5)),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(-100, 40, 180), 20.0), new OrenNayar(Color(0.5, 0.5, 0.5), 1.0),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(0, 40, 180), 20.0), new OrenNayar(Color(0.5, 0.5, 0.5), 0.5),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(-100, 40, 220), 20.0), new OrenNayar(Color(0.5, 0.5, 0.5), 0.5),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(-50, 40, 220), 20.0), new OrenNayar(Color(0.0, 0.5, 0.5), 0.2),
			   dev_geom, dev_shaders, dev_nodes);
	
	createNode(new Sphere(Vector(0, 40, 220), 20.0), new OrenNayar(Color(0.0, 0.0, 0.5), 0.9),
			   dev_geom, dev_shaders, dev_nodes);

	createNode(new Sphere(Vector(80, 40, 220), 20.0), new OrenNayar(Color(0.5, 0.0, 0.5), 0.9),
			   dev_geom, dev_shaders, dev_nodes);


}

__device__ 
Color raytrace(Ray ray, Geometry** dev_geom, Shader** dev_shaders, Node** dev_nodes)
{
	IntersectionData data;
	Node* closestNode = nullptr;

	data.dist = 1e99;
	
	for (int i = 0; i < GEOM_COUNT; i++)
	{
		if (dev_nodes[i]->geom->intersect(ray, data))
		{
			closestNode = dev_nodes[i];
		}
	}

	if (!closestNode)
	{
		return Color(0, 0, 0);
	}

	data.isVisible = testVisibility(dev_nodes, data);

	return closestNode->shader->shade(ray, data);
}

/**
 * The function checks if one of the red, green or blue components
 * of the colors a and b are too different.
 * @return true - if the difference is bigger than the THRESHOLD
 * @return false - if the difference is lower than the THRESHOLD
*/
__device__
bool tooDifferent(const Color& a, const Color& b)
{
	const float THRESHOLD = 0.1;
	return (fabs(a.r - b.r) > THRESHOLD ||
		     fabs(a.g - b.g) > THRESHOLD ||
		     fabs(a.b - b.b) > THRESHOLD);
}

__global__ 
void renderScene(Color* dev_vfb, Camera* dev_cam, Geometry** dev_geom, Shader** dev_shaders, Node** dev_nodes)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	
	if (offset < RES_X * RES_Y)
	{
		// first pass
		dev_vfb[offset] = raytrace(dev_cam->getScreenRay(x, y), dev_geom, dev_shaders, dev_nodes);
	
#ifdef ANTI_ALIASING
		const double kernel[5][2] = {
					{ 0, 0 },
					{ 0.3, 0.3 },
					{ 0.6, 0 },
					{ 0, 0.6 },
					{ 0.6, 0.6 },
				};
		
		// second pass
		Color neighs[5];
		neighs[0] = dev_vfb[offset];
		neighs[1] = dev_vfb[(x > 0 ? x - 1 : x) + y * blockDim.x * gridDim.x];
		neighs[2] = dev_vfb[(x + 1 < RES_X ? x + 1 : x) + y * blockDim.x * gridDim.x];
		neighs[3] = dev_vfb[x + (y > 0 ? y - 1 : y) * blockDim.x * gridDim.x];
		neighs[4] = dev_vfb[x + (y + 1 < RES_Y ? y + 1 : y) * blockDim.x * gridDim.x];

		Color average(0, 0, 0);
			
		for (int i = 0; i < 5; ++i)
		{
			average += neighs[i];
		}
		average /= 5.0f;
			
		for (int i = 0; i < 5; ++i)
		{
			if (tooDifferent(neighs[i], average))
			{
				needsAA[offset] = true;
				break;
			}
		}

		if (needsAA[offset])
		{
			Color result = dev_vfb[offset];
			
			for (int i = 1; i < 5; ++i)
			{
				result += raytrace(dev_cam->getScreenRay(x + kernel[i][0], y + kernel[i][1]), dev_geom, dev_shaders, dev_nodes);
			}
			dev_vfb[offset] = result / 5.0f;
		}
#endif
	}
}

/**
 * Wrapper kernel function
*/
extern "C" 
void cudaRenderer(Color* dev_vfb, Camera* dev_cam, Geometry** dev_geom, Shader** dev_shaders, Node** dev_nodes)
{
	initializeScene<<<1, 1>>>(dev_cam, dev_geom, dev_shaders, dev_nodes);

	dim3 THREADS_PER_BLOCK(32, 32); // 32*32 = 1024 (max threads per block supported)

	dim3 BLOCKS(RES_X / THREADS_PER_BLOCK.x, RES_Y / THREADS_PER_BLOCK.y); 

	renderScene<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_vfb, dev_cam, dev_geom, dev_shaders, dev_nodes);
}