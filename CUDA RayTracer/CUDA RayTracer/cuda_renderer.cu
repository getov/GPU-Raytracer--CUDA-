#include "hip/hip_runtime.h"
//#ifndef __HIPCC__  
//    #define __HIPCC__
//#endif
#include "hip/hip_runtime.h"
#include ""

#include "cuda_renderer.cuh"
#include "Vector3D.cuh"
#include "Matrix.cuh"
#include "Color.cuh"
#include "Camera.cuh"
#include "IGeometry.cuh"
#include "Plane.cuh"
#include "Node.cuh"
#include "IShader.cuh"
#include "Lambert.cuh"
#include "Sphere.cuh"
#include "OrenNayar.cuh"
#include "Phong.cuh"
#include "Refraction.cuh"
#include "Transform.cuh"
#include "Reflection.cuh"
#include "Layered.cuh"
#include "Fresnel.cuh"
#include "CameraController.cuh"
#include "RaytracerControls.cuh"
#include "Settings.cuh"
#include "WaterWaves.cuh"
//#include "custom_vector.cuh"
//
//using pgg::vector;

__device__
bool needsAA[RES_X * RES_Y];

__device__ 
Camera* dev_cam;

__device__ 
Geometry* dev_geom[GEOM_MAX_SIZE];

__device__ 
Shader* dev_shaders[GEOM_MAX_SIZE];

__device__ 
Node* dev_nodes[GEOM_MAX_SIZE];

__device__
Texture* dev_textures[GEOM_MAX_SIZE];

//__device__ vector<Geometry*> dev_geom;
//__device__ vector<Shader*> dev_shaders;
//__device__ vector<Node*> dev_nodes;
//__device__ vector<Texture*> dev_textures;
 
__device__
CameraController* m_controller;

__device__
bool testVisibility(const Vector& from, const Vector& to)
{
	Ray ray;
	ray.start = from;
	ray.dir = to - from;
	ray.dir.normalize();

	IntersectionData temp;
	temp.dist = (to - from).length();

	for (int i = 0; i < GEOM_COUNT; i++)
	{
		if (dev_nodes[i]->intersect(ray, temp))
		{
			return false;
		}
	}

	return true;
}

__device__
Node* createNode(Geometry* geom, Shader* shader, Texture* tex = nullptr)
{
	if (GEOM_COUNT >= GEOM_MAX_SIZE)
	{
		return;
	}
	
	dev_geom[GEOM_COUNT]     = geom;
	dev_shaders[GEOM_COUNT]  = shader;
	dev_textures[GEOM_COUNT] = tex;
	dev_nodes[GEOM_COUNT]    = new Node(dev_geom[GEOM_COUNT], dev_shaders[GEOM_COUNT], dev_textures[GEOM_COUNT]);

	return dev_nodes[GEOM_COUNT++];

	/*dev_geom.push_back(geom);
	dev_shaders.push_back(shader);
	dev_textures.push_back(tex);
	dev_nodes.push_back(new Node(geom, shader, tex));*/
}

__global__ 
void initializeScene()
{	
	dev_cam = new Camera;
	dev_cam->yaw = 0;
	dev_cam->pitch = 0;
	dev_cam->roll = 0;
	dev_cam->fov = 90;
	dev_cam->aspect = 4.0 / 3.0;
	dev_cam->pos = Vector(0, 150, -100);
	dev_cam->beginFrame();

	m_controller = new CameraController(*dev_cam, 10.f);
	
	lightPos = Vector(0, 296, 100);
	lightColor = Color(1, 1, 1);
	lightPower = 60000;
	ambientLight = Color(0.2, 0.2, 0.2);

#ifdef REAL_TIME_RENDERING

	//createNode(new Plane(5), new OrenNayar(Color(0.5, 0.5, 0.5), 1.0));

	////createNode(new Plane(500), new OrenNayar(Color(0.5, 0.5, 0.5), 1.0));

	//createNode(new Sphere(Vector(0, 50, 200), 40.0), new Phong(Color(0, 0, 1), 32));

	// ocean
	createNode(new Plane(-30), new Lambert(Color(0x0AB6FF)));  // 0.1448, 0.4742, 0.6804   0x0AB6FF
	Layered* water = new Layered;
	water->addLayer(new Refraction(Color(0.9, 0.9, 0.9), 1.33), Color(1.0, 1.0, 1.0));
	water->addLayer(new Reflection(Color(0.9, 0.9, 0.9)), Color(1.0, 1.0, 1.0), new Fresnel(1.33));
	
	Node* waterGeom = createNode(new Plane(0), water, new WaterWaves(0.2));
	waterGeom->transform.scale(5, 5, 5);

	createNode(new Sphere(Vector(10, -20, 250), 100.0), new Lambert(Color(0, 1, 0)));

#else

	/// room
	createNode(new Plane(5, 300, 300), new OrenNayar(Color(0xF5E08C), 1.0));

	Layered* mirror = new Layered;
	mirror->addLayer(new Reflection(), Color(1, 1, 1), new Fresnel(10.0));

	Node* BackWall = createNode(new Plane(-300, 300, 300), new OrenNayar(Color(0xF5E08C), 1.0));
	BackWall->transform.rotate(0, 90, 0);
	
	Node* SideWallLeft = createNode(new Plane(-150, 300, 300), new OrenNayar(Color(1.0, 0.0, 0.0), 1.0));
	SideWallLeft->transform.rotate(0, 0, 90);

	Node* SideWallRight = createNode(new Plane(150, 300, 300), new OrenNayar(Color(0.0, 0.0, 1.0), 1.0));
	SideWallRight->transform.rotate(0, 0, 90);

	Node* Roof = createNode(new Plane(300, 300, 300), new OrenNayar(Color(0xF5E08C), 1.0));

	Layered* moreGlossy = new Layered;
	moreGlossy->addLayer(new Phong(Color(0.0, 0.0, 1.0), 32), Color(1.0, 1.0, 1.0)); 
	moreGlossy->addLayer(new Reflection(Color(1.0, 1.0, 1.0)), Color(1, 1, 1), new Fresnel(2.5));
	createNode(new Sphere(Vector(0, 50, 200), 40.0), moreGlossy);

	Node* rectMirror = createNode(new Plane(0, 60, 80), mirror);
	rectMirror->transform.rotate(0, 90, 0);
	rectMirror->transform.translate(Vector(0, 120, 298));

	/// ocean
	//createNode(new Plane(-300), new Lambert(Color(0x0AB6FF))); 
	//Layered* water = new Layered;
	//water->addLayer(new Refraction(Color(0.9, 0.9, 0.9), 1.33), Color(1.0, 1.0, 1.0));
	//water->addLayer(new Reflection(Color(0.9, 0.9, 0.9)), Color(1.0, 1.0, 1.0), new Fresnel(1.33));
	//
	//Node* waterGeom = createNode(new Plane(0), water, new WaterWaves(0.2));
	//waterGeom->transform.scale(5, 5, 5);

	//createNode(new Sphere(Vector(50, -20, 350), 100.0), new Lambert(Color(0, 1, 0)));
	
#endif

}

__device__ 
Color raytrace(Ray ray)
{
	IntersectionData data;
	Node* closestNode = nullptr;

	if (ray.depth > MAX_RAY_DEPTH)
	{
		return Color(0, 0, 0);
	}

	data.dist = 1e99;

	for (int i = 0; i < GEOM_COUNT; ++i)
	{
		if (dev_nodes[i]->intersect(ray, data))
		{
			closestNode = dev_nodes[i];
		}
	}

	if (!closestNode)
	{
		//return Color(0, 0, 0);
		return Color(0.55f, 0.8f, 0.95f); // skyblue
		//return Color(1, 1, 1);
	}

	if (closestNode->bumpTex != nullptr)
	{
		closestNode->bumpTex->modifyNormal(data);
	}
	
	return closestNode->shader->shade(ray, data);
}

/**
 * @brief - The function checks if one of the red, green or blue components
 * of the colors a and b are too different.
 * @return true - if the difference is bigger than the THRESHOLD
 * @return false - if the difference is lower than the THRESHOLD
*/
__device__
bool tooDifferent(const Color& a, const Color& b)
{
	const float THRESHOLD = 0.1;
	return (fabs(a.r - b.r) > THRESHOLD ||
		     fabs(a.g - b.g) > THRESHOLD ||
		     fabs(a.b - b.b) > THRESHOLD);
}

__global__
void antiAliasing(Color* dev_vfb, bool previewAA)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;	
		
	const int n_size = 5;
	Color neighs[n_size];
	neighs[0] = dev_vfb[offset];
	neighs[1] = dev_vfb[(x > 0 ? x - 1 : x) + y * blockDim.x * gridDim.x];
	neighs[2] = dev_vfb[(x + 1 < RES_X ? x + 1 : x) + y * blockDim.x * gridDim.x];
	neighs[3] = dev_vfb[x + (y > 0 ? y - 1 : y) * blockDim.x * gridDim.x];
	neighs[4] = dev_vfb[x + (y + 1 < RES_Y ? y + 1 : y) * blockDim.x * gridDim.x];

	Color average(0, 0, 0);
			
	for (int i = 0; i < n_size; ++i)
	{
		average += neighs[i];
	}
	average /= static_cast<float>(n_size);
			
	for (int i = 0; i < n_size; ++i)
	{
		if (tooDifferent(neighs[i], average))
		{
			needsAA[offset] = true;
			break;
		}
	}

	const double kernel[5][2] = {
			{ 0, 0 },
			{ 0.3, 0.3 },
			{ 0.6, 0 },
			{ 0, 0.6 },
			{ 0.6, 0.6 },
		};

	if (previewAA)
	{
		if (needsAA[offset])
		{
			dev_vfb[offset] = Color(1, 0, 0);
		}
	}
	else
	{
		if (needsAA[offset])
		{
			Color result = dev_vfb[offset];
			
			for (int i = 1; i < n_size; ++i)
			{
				result += raytrace(dev_cam->getScreenRay(x + kernel[i][0], y + kernel[i][1]));
			}
			dev_vfb[offset] = result / static_cast<float>(n_size);
		}
	}
}

__global__ 
void renderScene(Color* dev_vfb)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < RES_X * RES_Y)
	{
		dev_vfb[offset] = raytrace(dev_cam->getScreenRay(x, y));
	}
}

__global__
void freeMemory()
{
	delete dev_cam;
	delete m_controller;

	for (int i = 0; i < GEOM_COUNT; ++i)
	{
		delete dev_geom[i];
		delete dev_shaders[i];
		delete dev_textures[i];
		delete dev_nodes[i];
	}

	printf("DELETED");
}

/**
 * Wrapper kernel function
*/
extern "C"
void initScene()
{
	initializeScene<<<1, 1>>>();
}

__global__
void camBeginFrame()
{
	dev_cam->beginFrame();
}

extern "C"
void cameraBeginFrame()
{
	camBeginFrame<<<1, 1>>>();
}

extern "C" 
void cudaRenderer(Color* dev_vfb)
{
	dim3 THREADS_PER_BLOCK(32, 32); // 32*32 = 1024 (max threads per block supported)
	dim3 BLOCKS(RES_X / THREADS_PER_BLOCK.x, RES_Y / THREADS_PER_BLOCK.y); 
	
	// first pass
	renderScene<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_vfb);

	//second pass
	if (GlobalSettings::AAEnabled)
	{
		antiAliasing<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_vfb, GlobalSettings::previewAA);
	}
}

extern "C"
void freeDeviceMemory()
{	
	freeMemory<<<1, 1>>>();
}