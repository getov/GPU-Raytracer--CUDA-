#include "hip/hip_runtime.h"
#include "Matrix.cuh"

__device__  void operator *= (Vector& v, const Matrix& a)
{
	v = v*a; 
}

__device__  Vector operator * (const Vector& v, const Matrix& m)
{
	return Vector(
		v.x * m.m[0][0] + v.y * m.m[1][0] + v.z * m.m[2][0],
		v.x * m.m[0][1] + v.y * m.m[1][1] + v.z * m.m[2][1],
		v.x * m.m[0][2] + v.y * m.m[1][2] + v.z * m.m[2][2]
	);
}

//!< matrix multiplication; result = a*b
__device__ Matrix operator * (const Matrix& a, const Matrix& b)
{
	Matrix c(0.0);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				c.m[i][j] += a.m[i][k] * b.m[k][j];
	return c;
}

//!< finds the determinant of a matrix
__device__ double determinant(const Matrix& a)
{
	return a.m[0][0] * a.m[1][1] * a.m[2][2]
	     - a.m[0][0] * a.m[1][2] * a.m[2][1]
	     - a.m[0][1] * a.m[1][0] * a.m[2][2]
	     + a.m[0][1] * a.m[1][2] * a.m[2][0]
	     + a.m[0][2] * a.m[1][0] * a.m[2][1]
	     - a.m[0][2] * a.m[1][1] * a.m[2][0];
}

__device__ double cofactor(const Matrix& m, int ii, int jj)
{
	int rows[2], rc = 0, cols[2], cc = 0;
	for (int i = 0; i < 3; i++)
		if (i != ii) rows[rc++] = i;
	for (int j = 0; j < 3; j++)
		if (j != jj) cols[cc++] = j;
	double t = m.m[rows[0]][cols[0]] * m.m[rows[1]][cols[1]] - m.m[rows[1]][cols[0]] * m.m[rows[0]][cols[1]];
	if ((ii + jj) % 2) t = -t;
	return t;
}

//!< finds the inverse of a matrix (assuming it exists)
__device__ Matrix inverseMatrix(const Matrix& m)
{
	double D = determinant(m);
	if (fabs(D) < 1e-12) return m; // an error; matrix is not invertible
	double rD = 1.0 / D;
	Matrix result;
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			result.m[i][j] = rD * cofactor(m, j, i);
	return result;
}

//!< returns a rotation matrix around the X axis; the angle is in radians
__device__ Matrix rotationAroundX(double angle)
{
	double S = sin(angle);
	double C = cos(angle);
	Matrix a(1.0);
	a.m[1][1] = C;
	a.m[2][1] = S;
	a.m[1][2] = -S;
	a.m[2][2] = C;
	return a;
}

//!< same as above, but rotate around Y
__device__ Matrix rotationAroundY(double angle)
{
	double S = sin(angle);
	double C = cos(angle);
	Matrix a(1.0);
	a.m[0][0] = C;
	a.m[2][0] = -S;
	a.m[0][2] = S;
	a.m[2][2] = C;
	return a;
}

//!< same as above, but rotate around Z
__device__ Matrix rotationAroundZ(double angle)
{
	double S = sin(angle);
	double C = cos(angle);
	Matrix a(1.0);
	a.m[0][0] = C;
	a.m[1][0] = S;
	a.m[0][1] = -S;
	a.m[1][1] = C;
	return a;
}