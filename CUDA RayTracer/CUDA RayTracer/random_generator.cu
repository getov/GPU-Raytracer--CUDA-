#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2009-2013 by Veselin Georgiev, Slavomir Kaslev et al    *
 *   admin@raytracing-bg.net                                               *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/
 
#include <math.h>
#include <SDL/SDL.h>
#include "random_generator.cuh"
#include "Settings.cuh"

__device__
Random::Random(unsigned seed)
{
	this->seed(seed);
}

__device__
void Random::seed(unsigned s)
{
	generator.seed(s);
}

__device__
unsigned Random::_next(void)
{
	std::uniform_int_distribution<unsigned> gen;
	return gen(generator);
}

__device__
int Random::randint(int a, int b)
{
	std::uniform_int_distribution<int> gen(a, b);
	return gen(generator);
}

__device__
float Random::randfloat(void)
{
	std::uniform_real_distribution<float> gen;
	return gen(generator);
}

__device__
double Random::randdouble(void)
{
	std::uniform_real_distribution<double> gen;
	return gen(generator);
}

__device__
double Random::gaussian(double mean, double sigma)
{
	std::normal_distribution<double> gen(mean, sigma);
	return gen(generator);
}

__device__
void Random::unitDiscSample(double &x, double &y)
{
	// pick a random point in the unit disc with uniform probability by using polar coords.
	// Note the sqrt(). For explanation why it's needed, see 
	// http://mathworld.wolfram.com/DiskPointPicking.html
	double angle = randdouble() * 2 * PI;
	double rad = sqrt(randdouble());
	x = sin(angle) * rad;
	y = cos(angle) * rad;
}

struct HashMapEntry {
	Random r;
	unsigned key;
	char fill[128]; // skip to the next cacheline
};

const int RGENS = 257; // 257 is a prime number
static HashMapEntry rg_table[RGENS];

__device__
void initRandom(unsigned seed)
{
	for (int i = 0; i < RGENS; i++)
		rg_table[i].key = 0xffffffff;
	const int MAXWARM = 1223;
	seed ^= 0xbf14ef80; // just in case the user passes '0'...
	// initialize and warm-up the zeroth random generator:
	rg_table[0].r.seed(seed);
	for (int i = 0; i < MAXWARM; i++) rg_table[0].r._next();
	for (int i = 1; i < RGENS; i++) {
		Random& prev = rg_table[i - 1].r;
		Random& next = rg_table[i].r;
		next.seed(prev._next());
		int n = prev.randint(0, MAXWARM - 1);
		for (int i = 0; i < n; i++)
			next._next();
	}
}

__device__
Random& getRandomGen(int idx)
{
	unsigned key = idx;
	int i = ((unsigned) idx % (unsigned) RGENS);
	for (int k = 0; k < RGENS; k++) {
		if (rg_table[i].key == key)
			return rg_table[i].r;
		else if (rg_table[i].key == 0xffffffff) {
			rg_table[i].key = key;
			return rg_table[i].r;
		} else {
			i++;
			if (i >= RGENS) i -= RGENS;
		}
	}
	return rg_table[i].r;
}

__device__
Random& getRandomGen()
{
	return getRandomGen(SDL_ThreadID());
}

// random generator testing code below (disabled)

#if 0

#include <time.h>
#include <stdio.h>
#include <SDL/SDL.h>
#include "constants.h"
#include "color.h"
#include "sdl.h"

extern Color vfb[VFB_MAX_SIZE][VFB_MAX_SIZE];

static Random* grand;

double genrandpair(void)
{
	double x, y;
	grand->unitDiscSample(x, y);
	return x;
}

double threadid(void)
{
	return SDL_ThreadID();
}

void testSpeed(const char* what, double (*fn)  (void))
{
	unsigned cntPerSec = 1;
	Uint32 c0, c1;
	int remaining = 5;
	bool useful = false;
	while (remaining) {
		double sum = 0;
		c0 = SDL_GetTicks();
		for (unsigned i = 0; i < cntPerSec; i++)
			sum += fn();
		c1 = SDL_GetTicks();
		double t = (c1 - c0)/1000.0;
		sum /= cntPerSec;
		if (useful) printf("%.2lf Million %s per second (avg = %.5lf)\n",
			cntPerSec/t/1000000.0, what, sum);
		if (t > 1) {
			remaining--;
			useful = true;
		} else {
			cntPerSec *= 2;
		}
		
	}
}

static int int_buff[512][512];
static float float_buff[512][512];
static float circle_buff[512][512];
static float norm_buff[512];

static void displayCharts(double npoints)
{
	
	for (int y = 0; y < 511; y++)
		for (int x = 0; x < 511; x++) {
			float f = int_buff[y][x] * 0.2f;
			vfb[y][x] = Color(f, f, f);
	}
	for (int y = 0; y < 511; y++)
		for (int x = 0; x < 512; x++) {
			float f = float_buff[y][x] * 0.2f;
			vfb[y][x+512] = Color(f, f, f);
		}
	for (int y = 0; y < 512; y++)
		for (int x = 0; x < 511; x++) {
			float f = circle_buff[y][x] * 0.2f;
			vfb[y+512][x] = Color(f, f, f);
		}
	const int BORDERS = 16;
	const int NLINES = 8;
	const int SPACING = (512 - 2 * BORDERS) / NLINES;
	double mult = (512 - 2 * BORDERS) * 512 * 0.4 / (npoints);
	for (int x = 0; x < 512; x++) {
		Color base = Color(0, 0, 0);
		if ((int) fabs(x - 256) % ((int) (256.0/3.0)) == 0)
			base = Color(0.2f, 0.2f, 0.2f);
		int ey = nearestInt(512 - BORDERS - norm_buff[x] * mult);
		int sy = 512 - BORDERS;
		for (int y = 0; y < 512; y++) {
			Color f = base;
			if (sy - y >= 0 && (sy - y) % SPACING == 0) f += Color(0.1f, 0.1f, 0.1f);
			if (y == ey) f = Color(0.4f, 0.4f, 1.0f);
			else if (y == sy) f = Color(0.9f, 0.9f, 0.9f);
			else if (y > sy) f.makeZero();
			else if (y > ey) f += Color(0.2f, 0.2f, 0.5f);
			vfb[y+512][x+512] = f;
		}
	}
	for (int i = 0; i < 1024; i++)
		vfb[i][511] = vfb[511][i] = Color(1, 1, 1);
}

void test_random()
{
	initGraphics(1024, 1024);
	Random rnd(time(NULL));
	//
	for (int i = 0; i < 200; i++)
		printf("mt: %u\n", rnd._next());
	
	SDL_WM_SetCaption("Testing MTRandom", NULL);
	
	grand = &rnd;
	
	testSpeed("unitDiscSample()s", genrandpair);
	
	testSpeed("SDL_ThreadID()s", threadid);
	
	double delayFactor = 1000.0;
	// test the random generator graphically:
	for (int i = 0; i < 30000; i++) {
		// generate new 1000 integer points:
		for (int j = 0; j < 1000; j++)
			int_buff[rnd.randint(0, 511)][rnd.randint(0, 511)]++;
		
		// generate new 1000 floatingpoint points:
		for (int j = 0; j < 1000; j++) {
			float x = rnd.randfloat()*512;
			float y = rnd.randfloat()*512;
			int x0 = (int) floor(x);
			int y0 = (int) floor(y);
			int x1 = (x0 + 1) % 512;
			int y1 = (y0 + 1) % 512;
			float p = x - x0;
			float q = y - y0;
			float_buff[y0][x0] += (1 - p) * (1 - q);
			float_buff[y0][x1] += (    p) * (1 - q);
			float_buff[y1][x0] += (1 - p) * (    q);
			float_buff[y1][x1] += (    p) * (    q);
		}
		
		// generate new 1000 circle points:
		for (int j = 0; j < 1000; j++) {
			double cx, cy;
			rnd.unitDiscSample(cx, cy);
			float x = float(cx * 256 + 256);
			float y = float(cy * 256 + 256);
			int x0 = (int) floor(x);
			int y0 = (int) floor(y);
			int x1 = (x0 + 1) % 512;
			int y1 = (y0 + 1) % 512;
			float p = x - x0;
			float q = y - y0;
			circle_buff[y0][x0] += (1 - p) * (1 - q);
			circle_buff[y0][x1] += (    p) * (1 - q);
			circle_buff[y1][x0] += (1 - p) * (    q);
			circle_buff[y1][x1] += (    p) * (    q);
		}
		
		// generate new 5000 normally-distributed points:
		for (int j = 0; j < 5000; j++) {
			double x = rnd.gaussian(256, 256.0/3.0);
			if (x < 0 || x >= 511) continue;
			double p = x - floor(x);
			int x0 = floor(x);
			norm_buff[x0] += (1 - p);
			norm_buff[x0 + 1] += p;
		}
		//
		displayCharts((i + 1) * 5000);
		displayVFB(vfb);
		SDL_Delay((int) delayFactor);
		delayFactor *= 0.9;
	}
	waitForUserExit();
	closeGraphics();
}
#endif
