#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""

#include <SDL/SDL.h>
#include <iostream>
#include "cuda_renderer.cuh"
#include "Vector3D.cuh"
#include <vector>
#include "sdl.cuh"
#include "Color.cuh"
#include "Camera.cuh"
#include "Matrix.cuh"
#include "IGeometry.cuh"
#include "IShader.cuh"
#include "Node.cuh"
#include "Lambert.cuh"
#include "Plane.cuh"
	
using namespace std;

extern "C" void cudaRenderer(Color* dev_vfb, Camera* dev_cam, Geometry** dev_geom, Shader** dev_shaders, Node** dev_nodes);

Color vfb[RES_X][RES_Y];

// used for GPU operations
Color vfb_linear[RES_X * RES_Y]; 

//const int ARR_SIZE = 3;

Camera* camera;
Geometry* geometry[GEOM_COUNT];
Shader* shaders[GEOM_COUNT];
Node* nodes[GEOM_COUNT];

void printGPUSpecs()
{
	hipDeviceProp_t  prop;
    int count;
    hipGetDeviceCount(&count);
	//printf( "%d", sizeof(nodes)/sizeof(Node*));
    for (int i = 0; i < count; ++i) 
	{
        hipGetDeviceProperties( &prop, i );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );

        if (prop.deviceOverlap)
		{
            printf( "Enabled\n" );
		}
        else
		{
            printf( "Disabled\n");
		}
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
		{
            printf( "Enabled\n" );
		}
        else
		{
            printf( "Disabled\n" );
		}

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
    }
}

void cudaStartTimer(hipEvent_t& start, hipEvent_t& stop)
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void cudaStopTimer(hipEvent_t& start, hipEvent_t& stop)
{
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float  elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to render:  %3.1f ms\n\n", elapsedTime);
	
	hipEventDestroy(start);
    hipEventDestroy(stop);
}

void convertDeviceToHostBuffer()
{
	for (int i = 0; i < RES_X; ++i)
	{
		for (int j = 0; j < RES_Y; ++j)
		{
			vfb[i][j] = vfb_linear[i * RES_X + j];
		}
	}
}

int main(int argc, char** argv)
{
	if (!initGraphics(RES_X, RES_Y))
	{
		return -1;
	}

	// capture the start time
	hipEvent_t start, stop;
	cudaStartTimer(start, stop);

	// 1. allocate memory for vfb on the GPU
	Color* dev_vfb;
	hipMalloc((void**)&dev_vfb, sizeof(Color) * RES_X * RES_Y);

	Camera* dev_cam;
	hipMalloc((void**)&dev_cam, sizeof(Camera));

	Geometry** dev_geom;
	hipMalloc((void**)&dev_geom, sizeof(Geometry) * GEOM_COUNT);

	Shader** dev_shaders;
	hipMalloc((void**)&dev_shaders, sizeof(Shader) * GEOM_COUNT);

	Node** dev_nodes;
	hipMalloc((void**)&dev_nodes, sizeof(Node) * GEOM_COUNT);
	
	// 2. memcpy HostToDevice
	hipMemcpy(dev_vfb, vfb_linear, sizeof(Color) * RES_X * RES_Y, hipMemcpyHostToDevice);
	hipMemcpy(dev_cam, camera, sizeof(Camera), hipMemcpyHostToDevice);
	hipMemcpy(dev_geom, geometry, sizeof(Geometry) * GEOM_COUNT, hipMemcpyHostToDevice);
	hipMemcpy(dev_shaders, shaders, sizeof(Shader) * GEOM_COUNT, hipMemcpyHostToDevice);
	hipMemcpy(dev_nodes, nodes, sizeof(Node) * GEOM_COUNT, hipMemcpyHostToDevice);

	// 3. call kernels
	// - InitializeScene
	// - RenderScene
	cudaRenderer(dev_vfb, dev_cam, dev_geom, dev_shaders, dev_nodes);

	// 4. memcpy DeviceToHost
	hipMemcpy(vfb_linear, dev_vfb, sizeof(Color) * RES_X * RES_Y, hipMemcpyDeviceToHost);

	// get stop time, and display the timing results
	cudaStopTimer(start, stop);

	printGPUSpecs();

	// 5. free memory
	hipFree(dev_vfb);
	hipFree(dev_cam);
	hipFree(dev_geom);
	hipFree(dev_shaders);
	hipFree(dev_nodes);

	// convert the linear array to our 2D array
	convertDeviceToHostBuffer();
	
	displayVFB(vfb);

	waitForUserExit();
	
	closeGraphics();

	return EXIT_SUCCESS;
}