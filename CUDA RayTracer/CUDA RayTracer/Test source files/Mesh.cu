#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2009-2013 by Veselin Georgiev, Slavomir Kaslev et al    *
 *   admin@raytracing-bg.net                                               *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/

#include <stdio.h>
#include <string.h>
#include <algorithm>
#include <string>
#include <vector>
#include "Mesh.cuh"
#include "Settings.cuh"
#include "Color.cuh"
#include "AABB.cuh"
#include "Util.cuh"

using std::string;

__device__
void Mesh::initMesh(void)
{
	// calculate a bounding box around the mesh:
	boundingBox.makeEmpty();
	for (int i = 1; i < (int) vertices.size(); i++)
		boundingBox.add(vertices[i]);
	kdroot = NULL;
	if (triangles.size() > 40 && useKDTree) {
		kdroot = new KDTreeNode;
		pgg::vector<int> allTriangles;
		for (int i = 0; i < (int) triangles.size(); i++)
			allTriangles.push_back(i);
		build(*kdroot, boundingBox, allTriangles, 0);
	}
}

__device__
Mesh::~Mesh()
{
	if (kdroot) delete kdroot;
}

//__device__
//const char* Mesh::getName()
//{
//	static char temp[200];
//	sprintf(temp, "Mesh with %d vertices, %d triangles\n", (int) vertices.size(), (int) triangles.size());
//	return temp;
//}

__device__
bool intersectTriangleFast(const Ray& ray, const Vector& A, const Vector& B, const Vector& C, double& dist)
{
	Vector AB = B - A;
	Vector AC = C - A;
	Vector D = -ray.dir;
	//              0               A
	Vector H = ray.start - A;

	/* 2. Solve the equation:
	 *
	 * A + lambda2 * AB + lambda3 * AC = ray.start + gamma * ray.dir
	 *
	 * which can be rearranged as:
	 * lambda2 * AB + lambda3 * AC + gamma * D = ray.start - A
	 *
	 * Which is a linear system of three rows and three unknowns, which we solve using Carmer's rule
	 */

	// Find the determinant of the left part of the equation:
	double Dcr = (AB ^ AC) * D;
	
	// are the ray and triangle parallel?
	if (fabs(Dcr) < 1e-12) return false;
	
	double lambda2 = ( ( H ^ AC) * D ) / Dcr;
	double lambda3 = ( (AB ^  H) * D ) / Dcr;
	double gamma   = ( (AB ^ AC) * H ) / Dcr;

	// is intersection behind us, or too far?
	if (gamma < 0 || gamma > dist) return false;

	// is the intersection outside the triangle?
	if (lambda2 < 0 || lambda2 > 1 || lambda3 < 0 || lambda3 > 1 || lambda2 + lambda3 > 1)
		return false;

	dist = gamma;
	
	
	return true;
}


__device__
bool Mesh::intersectTriangle(const Ray& ray, IntersectionData& data, Triangle& T)
{
	bool inSameDirection = (dot(ray.dir, T.gnormal) > 0);
	//if (backfaceCulling && inSameDirection && !(ray.flags & RF_SHADOW)) return false; // backface culling
	// (backface culling needs to be disabled when we trace shadow rays, otherwise we may find light
	//  in places there shouldn't be one).
	//              B                     A
	Vector AB = vertices[T.v[1]] - vertices[T.v[0]];
	Vector AC = vertices[T.v[2]] - vertices[T.v[0]];
	Vector D = -ray.dir;
	//              0               A
	Vector H = ray.start - vertices[T.v[0]];

	/* 2. Solve the equation:
	 *
	 * A + lambda2 * AB + lambda3 * AC = ray.start + gamma * ray.dir
	 *
	 * which can be rearranged as:
	 * lambda2 * AB + lambda3 * AC + gamma * D = ray.start - A
	 *
	 * Which is a linear system of three rows and three unknowns, which we solve using Carmer's rule
	 */

	// Find the determinant of the left part of the equation:
	double Dcr = (AB ^ AC) * D;
	
	// are the ray and triangle parallel?
	if (fabs(Dcr) < 1e-12) return false;
	
	double rDcr = 1/Dcr;
	
	double lambda2 = ( ( H ^ AC) * D ) * rDcr;
	double lambda3 = ( (AB ^  H) * D ) * rDcr;
	double gamma   = ( (AB ^ AC) * H ) * rDcr;

	// is intersection behind us, or too far?
	if (gamma < 0 || gamma > data.dist) return false;
	
	// is the intersection outside the triangle?
	if (lambda2 < 0 || lambda2 > 1 || lambda3 < 0 || lambda3 > 1 || lambda2 + lambda3 > 1)
		return false;
	//
	
	// intersection found, and it's closer to the current one in data.
	// store intersection point.
	data.p = ray.start + ray.dir * gamma;
	data.dist = gamma;
	data.g = this;
	
	double lambda1 = 1 - lambda2 - lambda3;
	if (faceted || !hasNormals) {
		data.normal = T.gnormal;
	} else {
		// interpolate normals using the barycentric coords:
		data.normal = normals[T.n[0]] * lambda1 +
					  normals[T.n[1]] * lambda2 +
					  normals[T.n[2]] * lambda3;
		data.normal.normalize();
	}
	
	// interpolate the UV texture coords using barycentric coords:
	Vector uv = uvs[T.t[0]] * lambda1 +
				uvs[T.t[1]] * lambda2 +
				uvs[T.t[2]] * lambda3;
	data.u = uv.x;
	data.v = uv.y;
	data.dNdx = T.dNdx;
	data.dNdy = T.dNdy;
	return true;
}

__device__
bool Mesh::intersectKD(KDTreeNode& node, const BBox& bbox, const Ray& ray, IntersectionData& data)
{
	if (node.axis == AXIS_NONE) {
		// leaf node; try intersecting with the triangle list:
		bool found = false;
		for (size_t i = 0; i < node.triangles->size(); i++) {
			int triIdx = (*node.triangles)[i];
			if (intersectTriangle(ray, data, triangles[triIdx])) {
				found = true;
			}
		}
		// the found intersection has to be inside "our" BBox, otherwise we can miss a triangle,
		// as explained in the presentations:
		if (found && bbox.inside(data.p)) return true;
		return false;
	} else {
		// a in-node; intersect with the two children, starting with the closer one first:
		int childOrder[2] = { 0, 1 };
		if (ray.start[node.axis] > node.splitPos)
			dev_swap(childOrder[0], childOrder[1]);
		// 
		BBox childBB[2];
		bbox.split(node.axis, node.splitPos, childBB[0], childBB[1]);
		// name the children bboxen:
		BBox& firstBB = childBB[childOrder[0]];
		BBox& secondBB = childBB[childOrder[1]];
		KDTreeNode& firstChild = node.children[childOrder[0]];
		KDTreeNode& secondChild = node.children[childOrder[1]];
		// if the ray intersects the common wall between the two sub-boxes, then it invariably
		// intersects both boxes (we can skip the testIntersect() checks):
		// (see http://raytracing-bg.net/?q=node/68 )
		if (bbox.intersectWall(node.axis, node.splitPos, ray)) {
			if (intersectKD(firstChild, firstBB, ray, data)) return true;
			return intersectKD(secondChild, secondBB, ray, data);
		} else {
			// if the wall isn't hit, then we intersect exclusively one of the sub-boxes;
			// test one, if the test fails, then it's in the other:
			if (firstBB.testIntersect(ray))
				return intersectKD(firstChild, firstBB, ray, data);
			else
				return intersectKD(secondChild, secondBB, ray, data);
		}
	}
}

__device__
bool Mesh::intersect(Ray ray, IntersectionData& data)
{
	bool found = false;
	// if the ray doesn't intersect the bounding shpere, it is of no use
	// to continue: it can't possibly intersect the mesh.
	if (!boundingBox.testIntersect(ray)) return false;
	
	// if we built a KDTree, use that:
	if (kdroot) {
		return intersectKD(*kdroot, boundingBox, ray, data);
	} else {
		// naive algorithm - iterate and check for intersection all triangles:
		for (size_t i = 0; i < triangles.size(); i++) {
			if (intersectTriangle(ray, data, triangles[i]))
				found = true;
		}
		return found;
	}
}

// parse a string, convert to double. If string is empty, return 0
//__device__
//static double getDouble(const char* s)
//{
//	c_string tmp;
//
//	return static_cast<double>(tmp.atoi(s));
//}
//
//// parse a string, convert to int. If string is empty, return 0
//__device__
//static int getInt(const char* s)
//{
//	c_string tmp;
//	return tmp.atoi(s);
//}

// create a triangle by a OBJ file "f"-line, like "f 1//3 5//3 6//3". The three params in this
// case will be "1//3", "5//3" and "6//3"
//__device__
//Triangle::Triangle(const char* a, const char* b, const char* c)
//{
//	const char* items[3] = { a, b, c };
//	
//	for (int i = 0; i < 3; i++) {
//		const char* item = items[i];
//		
//		vector<char*> subItems = split(item, '/');
//		v[i] = getInt(subItems[0]);
//		if (subItems.size() > 1) {
//			t[i] = getInt(subItems[1]);
//		} else t[i] = 0;
//		if (subItems.size() > 2) {
//			n[i] = getInt(subItems[2]);
//		} else n[i] = 0;
//	}
//}

Triangle::Triangle(std::string a, std::string b, std::string c)
{
	string items[3] = { a, b, c };
	
	for (int i = 0; i < 3; i++) {
		const string& item = items[i];
		
		std::vector<string> subItems = split(item, '/');
		v[i] = getInt(subItems[0]);
		if (subItems.size() > 1) {
			t[i] = getInt(subItems[1]);
		} else t[i] = 0;
		if (subItems.size() > 2) {
			n[i] = getInt(subItems[2]);
		} else n[i] = 0;
	}
}

__device__
void solve2D(double M[2][2], double H[2], double& p, double& q)
{
	// solve a 2x2 linear system:
	// (p, q) * (M) = (H)
	// where p, q are scalars ("unknowns"), M is a 2x2 matrix, and H is a 2-tuple.
	
	double Dcr = M[0][0] * M[1][1] - M[1][0] * M[0][1];
	
	double rDcr = 1 / Dcr;
	
	p = (H[0] * M[1][1] - H[1] * M[0][1]) * rDcr;
	q = (M[0][0] * H[1] - M[1][0] * H[0]) * rDcr;
}


__device__ 
bool Mesh::loadFromOBJ()
{
	//FILE* f = fopen(filename, "rt");

	//if (!f) {
	//	printf("error: no such file: %s", filename);
	//	return false;
	//}
	//
	//vertices.push_back(Vector(0, 0, 0));
	//uvs.push_back(Vector(0, 0, 0));
	//normals.push_back(Vector(0, 0, 0));
	//hasNormals = false;
	//
	//
	//char line[2048];
	//
	//while (fgets(line, sizeof(line), f)) {
	//	if (line[0] == '#') continue;
	//	
	//	vector<char*> tokens = tokenize(line);
	//	
	//	if (tokens.size() == 0) continue;
	//	
	//	// v line - a vertex definition
	//	if (tokens[0] == "v") {
	//		Vector t(getDouble(tokens[1]),
	//		         getDouble(tokens[2]),
	//		         getDouble(tokens[3]));
	//		vertices.push_back(t);
	//		continue;
	//	}

	//	// vn line - a vertex normal definition
	//	if (tokens[0] == "vn") {
	//		hasNormals = true;
	//		Vector t(getDouble(tokens[1]),
	//		         getDouble(tokens[2]),
	//		         getDouble(tokens[3]));
	//		normals.push_back(t);
	//		continue;
	//	}

	//	// vt line - a texture coordinate definition
	//	if (tokens[0] == "vt") {
	//		Vector t(getDouble(tokens[1]),
	//		         getDouble(tokens[2]),
	//		         0);
	//		uvs.push_back(t);
	//		continue;
	//	}
	//	
	//	// f line - a face definition
	//	if (tokens[0] == "f") {
	//		int numTriangles = tokens.size() - 3;
	//		
	//		for (int i = 0; i < numTriangles; i++) {
	//			Triangle T(tokens[1], tokens[2 + i], tokens[3 + i]);
	//			triangles.push_back(T);
	//		}
	//	}
	//}
	
	// preprocess all triangles:
	for (int i = 0; i < (int) triangles.size(); i++) {
		Triangle& T = triangles[i];
		
		Vector AB = vertices[T.v[1]] - vertices[T.v[0]];
		Vector AC = vertices[T.v[2]] - vertices[T.v[0]];
		
		// compute the geometric normal of this triangle:
		T.gnormal = AB ^ AC;
		T.gnormal.normalize();
		
		
		// compute the dNd(x|y) vectors of this triangle:
		double px, py, qx, qy;
		
		Vector AB_2d = uvs[T.t[1]] - uvs[T.t[0]];
		Vector AC_2d = uvs[T.t[2]] - uvs[T.t[0]];
		
		double mat[2][2] = {
			{ AB_2d.x, AC_2d.x },
			{ AB_2d.y, AC_2d.y },
		};
		double h[2] = { 1, 0 };
		
		solve2D(mat, h, px, qx); // (AB_2d * px + AC_2d * qx == (1, 0))
		h[0] = 0; h[1] = 1;
		solve2D(mat, h, py, qy); // (AB_2d * py + AC_2d * qy == (0, 1))
		
		T.dNdx = AB * px + AC * qx;
		T.dNdx.normalize();
		T.dNdy = AB * py + AC * qy;
		T.dNdy.normalize();
	}
	// create the normals[] array - if needed:
	if (!hasNormals && autoSmooth) {
		hasNormals = true;
		normals.resize(vertices.size(), Vector(0, 0, 0)); // extend the normals[] array, and fill with zeros
		for (int i = 0; i < (int) triangles.size(); i++)
			for (int j = 0; j < 3; j++) {
				triangles[i].n[j] = triangles[i].v[j];
				normals[triangles[i].n[j]] += triangles[i].gnormal;
			}
		for (int i = 1; i < (int) normals.size(); i++)
			if (normals[i].lengthSqr() > 1e-9) normals[i].normalize();
	}

	//fclose(f);
	return true;
}

__device__
void Mesh::build(KDTreeNode& node, const BBox& bbox, pgg::vector<int>& tList, int depth)
{
	if (tList.size() < MAX_TRIANGLES_PER_LEAF || depth > MAX_TREE_DEPTH) {
		node.initLeaf(tList);
	} else {
		Axis axis = (Axis) (depth % 3); // alternate splitting planes: X, Y, Z, X, Y, Z, ...
		double axisL = bbox.vmin[axis]; // the left and right extents of the bbox along the chosen axis
		double axisR = bbox.vmax[axis];
		
		// naive split-position choice here: just use the middle of the current bbox.
		// A smarter algo could be used here:
		double splitPos = (axisL + axisR) * 0.5;
		BBox bbLeft, bbRight;
		bbox.split(axis, splitPos, bbLeft, bbRight);
		
		// Split the triangle list into tLeft, tRight, depending on which BBox the triangles
		// intersect with.
		pgg::vector<int> tLeft, tRight;
		for (int i = 0; i < (int) tList.size(); i++) {
			Triangle& T = triangles[tList[i]];
			const Vector& A = vertices[T.v[0]];
			const Vector& B = vertices[T.v[1]];
			const Vector& C = vertices[T.v[2]];
			// usually, a triangle will go either in the left or the right list. In some
			// cases, it may go in both (which is bad, but we hope this would be rare):
			if (bbLeft.intersectTriangle(A, B, C))
				tLeft.push_back(tList[i]);
			if (bbRight.intersectTriangle(A, B, C))
				tRight.push_back(tList[i]);
		}
		node.initBinary(axis, splitPos);
		build(node.children[0], bbLeft, tLeft, depth + 1);
		build(node.children[1], bbRight, tRight, depth + 1);
	}
}
