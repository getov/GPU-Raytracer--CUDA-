#include "hip/hip_runtime.h"
#include "CameraController.cuh"

__device__
CameraController::CameraController(Camera& camera, const float& speed)
	: m_camera(&camera)
	, movementSpeed(speed)
	, q(0.0)
	, n(1.0)
{
}

__device__ 
Vector CameraController::forward()
{
	return m_camera->frontDir;
}

__device__ 
Vector CameraController::backward()
{
	return -m_camera->frontDir;
}

__device__ 
Vector CameraController::right()
{
	return m_camera->rightDir;
}

__device__
Vector CameraController::left()
{
	return -m_camera->rightDir;
}

__device__ 
void CameraController::moveForward()
{
	m_camera->pos += forward() * movementSpeed * scene->secondsElapsed;
}

__device__
void CameraController::moveBackward()
{
	m_camera->pos += backward() * movementSpeed * scene->secondsElapsed;
}

__device__ 
void CameraController::strafeRight()
{
	m_camera->pos += right() * movementSpeed * scene->secondsElapsed;
}

__device__ 
void CameraController::strafeLeft()
{
	m_camera->pos += left() * movementSpeed * scene->secondsElapsed;
}

// rotation
__device__                         
void CameraController::offsetCameraOrientation(const float& zenith, const float& azimuth)
{
	m_camera->yaw += -azimuth;

	while (m_camera->yaw > 360.0f)
	{
		m_camera->yaw -= 360.0;
	}
	while (m_camera->yaw < 0.0f)
	{
		m_camera->yaw += 360.0;
	}

	// to prevent gimbal lock
	m_camera->pitch += -zenith;
	if (m_camera->pitch > 85.0f)
	{
		m_camera->pitch = 85.0f;
	}
	if (m_camera->pitch < -85.0f)
	{
		m_camera->pitch = -85.0f;
	}
}

__device__
void CameraController::quaternion()
{
	q += n * scene->secondsElapsed;

	if (q > PI/12)
	{
		q = PI/12;
		n *= -1.0;
	}
	else if (q < -PI/12)
	{
		q = -PI/12;
		n *= -1.0;
	}

	m_camera->roll = toDegrees(asin(sin(q)));
}